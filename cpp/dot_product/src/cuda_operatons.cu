#include "cuda_util.h"
#include "common.h"
#include "hip/hip_fp16.h"

void runCudaOperations(float* a, float* b, float* out, size_t a_size, size_t b_size, size_t out_size, unsigned int ilen, unsigned int alen, unsigned int blen) {
    float *d_a, *d_b, *d_out;
    hipMalloc(&d_a, a_size);
    hipMalloc(&d_b, b_size);
    hipMalloc(&d_out, out_size);

    hipMemcpy(d_a, a, a_size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, b_size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((alen + 15) / 16, (blen + 15) / 16);

    // start measuring time of computing
    hipEvent_t start, stop;
    float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    //dotProductKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_out, ilen, alen, blen);
    dotProductKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_out, ilen, alen, blen);
    hipDeviceSynchronize();

    // stop measuring time
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Kernel execution time: " << milliseconds << " milliseconds.\n";

    hipError_t error = hipGetLastError();
    checkCudaError(error);

    hipMemcpy(out, d_out, out_size, hipMemcpyDeviceToHost);

    error = hipMalloc(&d_a, a_size);
    checkCudaError(error);

    error = hipMemcpy(d_a, a, a_size, hipMemcpyHostToDevice);
    checkCudaError(error);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
}

void getSystemInformation() {
    int device;
    hipGetDevice(&device); // get current device ID
    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, device); // get device properties
    std::cout << "GPU Name: " << properties.name << std::endl;
    std::cout << "Compute capability: " << properties.major << "." << properties.minor << std::endl;
    std::cout << "Total global memory: " << properties.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
    std::cout << "Shared memory per block: " << properties.sharedMemPerBlock / 1024 << " KB" << std::endl;
    std::cout << "Registers per block: " << properties.regsPerBlock << std::endl;
    std::cout << "Warp size: " << properties.warpSize << std::endl;
    std::cout << "Memory Clock Rate (KHz): " << properties.memoryClockRate << std::endl;
    std::cout << "Memory Bus Width (bits): " << properties.memoryBusWidth << std::endl;
    if (properties.memoryBusWidth != 0) {
        float memoryBandwidth = 2.0 * properties.memoryClockRate * (properties.memoryBusWidth / 8) / 1.0e6;
        std::cout << "Theoretical Memory Bandwidth (GB/s): " << memoryBandwidth << std::endl;
    }
}