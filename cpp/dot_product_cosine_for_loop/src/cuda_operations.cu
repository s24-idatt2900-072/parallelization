#include "hip/hip_runtime.h"
#include "cuda_util.h"
#include "common.h"
#include "hip/hip_fp16.h"


#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            std::cerr << "Error: " << hipGetErrorString(error) << ", file: " << __FILE__ << ", line: " << __LINE__ << std::endl; \
            exit(1); \
        } \
    } while (0)

void runCosineSimiliarity(float* images, float* filter_real, float* filter_abs, float* output, size_t images_size, size_t images_vector_len, size_t real_vector_len, size_t filters_size, size_t output_size, unsigned int inner_len, unsigned int image_len, unsigned int filter_len, size_t &memory_used_MB_dot_product, size_t &memory_free_MB_dot_product) {
    float *d_images, *d_filter_real, *d_filter_abs, *d_output;
    size_t free_before, total_before, free_after, total_after;

    CUDA_CHECK(hipMemGetInfo(&free_before, &total_before));

    CUDA_CHECK(hipMalloc(&d_images, images_size));
    CUDA_CHECK(hipMalloc(&d_filter_real, filters_size));
    CUDA_CHECK(hipMalloc(&d_filter_abs, filters_size));
    CUDA_CHECK(hipMalloc(&d_output, output_size));

    CUDA_CHECK(hipMemcpy(d_images, images, images_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_filter_real, filter_real, filters_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_filter_abs, filter_abs, filters_size, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(16, 16, 1);
    dim3 blocksPerGrid((image_len + 15) / 16, (filter_len + 15) / 16);

    cosineSimilarityKernel<<<blocksPerGrid, threadsPerBlock>>>(d_images, d_filter_real, d_filter_abs, d_output, inner_len, image_len, images_vector_len, real_vector_len, filter_len);
    hipDeviceSynchronize();

    CUDA_CHECK(hipMemGetInfo(&free_after, &total_after));

    memory_used_MB_dot_product = (free_before - free_after) / 1024 / 1024;
    memory_free_MB_dot_product = free_after / 1024 / 1024;

    CUDA_CHECK(hipMemcpy(output, d_output, output_size, hipMemcpyDeviceToHost));

    hipFree(d_images);
    hipFree(d_filter_real);
    hipFree(d_filter_abs);
    hipFree(d_output);
}



void runMaxPool(float* output, float* pooled_output, size_t output_size, unsigned int pool_size, unsigned int pool_len, size_t &memory_used_MB_max_pool, size_t &memory_free_MB_max_pool) {
    float *d_output, *d_pooled_output;
    size_t free_before, total_before, free_after, total_after;

    CUDA_CHECK(hipMemGetInfo(&free_before, &total_before));

    CUDA_CHECK(hipMalloc(&d_output, output_size));
    CUDA_CHECK(hipMalloc(&d_pooled_output, pool_len * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_output, output, output_size, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(256, 1, 1);
    dim3 blocksPerGrid((pool_len + threadsPerBlock.x - 1) / threadsPerBlock.x, 1, 1);

    maxPoolKernel<<<blocksPerGrid, threadsPerBlock>>>(d_output, d_pooled_output, pool_size, pool_len);
    hipDeviceSynchronize();

    CUDA_CHECK(hipMemGetInfo(&free_after, &total_after));

    memory_used_MB_max_pool = (free_before - free_after) / 1024 / 1024;
    memory_free_MB_max_pool = free_after / 1024 / 1024;

    CUDA_CHECK(hipMemcpy(pooled_output, d_pooled_output, pool_len * sizeof(float), hipMemcpyDeviceToHost));

    hipFree(d_output);
    hipFree(d_pooled_output);
}

void runCombinedOperations(
    float* images, float* filter_real, float* filter_abs, float* pooled_output,
    size_t images_size, size_t images_vector_len, size_t real_vector_len, size_t filters_size, size_t output_size, size_t pooled_output_size,
    unsigned int inner_len, unsigned int image_len, unsigned int filter_len, unsigned int pool_size, unsigned int pool_len,
    size_t &memory_used, size_t &memory_free) {

    float *d_images, *d_filter_real, *d_filter_abs, *d_output, *d_pooled_output;
    size_t free_before, total_before, free_after, total_after;

    // Get initial memory status
    CUDA_CHECK(hipMemGetInfo(&free_before, &total_before));

    // Allocate memory on device
    CUDA_CHECK(hipMalloc(&d_images, images_size));
    CUDA_CHECK(hipMalloc(&d_filter_real, filters_size));
    CUDA_CHECK(hipMalloc(&d_filter_abs, filters_size));
    CUDA_CHECK(hipMalloc(&d_output, output_size));
    CUDA_CHECK(hipMalloc(&d_pooled_output, pooled_output_size));

    // Copy data from host to device
    CUDA_CHECK(hipMemcpy(d_images, images, images_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_filter_real, filter_real, filters_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_filter_abs, filter_abs, filters_size, hipMemcpyHostToDevice));

    dim3 threadsPerBlockCosine(16, 16, 1);
    dim3 blocksPerGridCosine((image_len + 15) / 16, (filter_len + 15) / 16);

    cosineSimilarityKernel<<<blocksPerGridCosine, threadsPerBlockCosine>>>(d_images, d_filter_real, d_filter_abs, d_output, inner_len, image_len, images_vector_len, real_vector_len, filter_len);
    CUDA_CHECK(hipDeviceSynchronize());

    // Define grid and block sizes for pooling
    dim3 threadsPerBlockPool(256, 1, 1);
    dim3 blocksPerGridPool((pool_len + threadsPerBlockPool.x - 1) / threadsPerBlockPool.x, 1, 1);

    // Run max pooling kernel
    maxPoolKernel<<<blocksPerGridPool, threadsPerBlockPool>>>(d_output, d_pooled_output, pool_size, pool_len);
    CUDA_CHECK(hipDeviceSynchronize());

    // Get final memory status
    CUDA_CHECK(hipMemGetInfo(&free_after, &total_after));

    // Calculate memory used
    memory_used = (free_before - free_after) / 1024 / 1024;
    memory_free = free_after / 1024 / 1024;

    // Copy results back to host
    CUDA_CHECK(hipMemcpy(pooled_output, d_pooled_output, pooled_output_size, hipMemcpyDeviceToHost));

    // Cleanup
    hipFree(d_images);
    hipFree(d_filter_real);
    hipFree(d_filter_abs);
    hipFree(d_output);
    hipFree(d_pooled_output);
}


void getSystemInformation() {
    int device;
    hipGetDevice(&device); // get current device ID
    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, device); // get device properties
    std::cout << "GPU Name: " << properties.name << std::endl;
    std::cout << "Compute capability: " << properties.major << "." << properties.minor << std::endl;
    std::cout << "Total global memory: " << properties.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
    std::cout << "Shared memory per block: " << properties.sharedMemPerBlock / 1024 << " KB" << std::endl;
    std::cout << "Registers per block: " << properties.regsPerBlock << std::endl;
    std::cout << "Warp size: " << properties.warpSize << std::endl;
    std::cout << "Memory Clock Rate (KHz): " << properties.memoryClockRate << std::endl;
    std::cout << "Memory Bus Width (bits): " << properties.memoryBusWidth << std::endl;
    if (properties.memoryBusWidth != 0) {
        float memoryBandwidth = 2.0 * properties.memoryClockRate * (properties.memoryBusWidth / 8) / 1.0e6;
        std::cout << "Theoretical Memory Bandwidth (GB/s): " << memoryBandwidth << std::endl;
    }

    size_t free_byte;
    size_t total_byte;
    hipError_t cuda_status = hipMemGetInfo(&free_byte, &total_byte);

    if (hipSuccess != cuda_status) {
        std::cout << "Error: hipMemGetInfo fails, " << hipGetErrorString(cuda_status) << std::endl;
        exit(1);
    }

    double free_db = (double)free_byte;
    double total_db = (double)total_byte;
    double used_db = total_db - free_db;
    std::cout << "GPU memory usage: used = " << used_db / 1024.0 / 1024.0 << " MB, free = " << free_db / 1024.0 / 1024.0 << " MB, total = " << total_db / 1024.0 / 1024.0 << " MB" << std::endl;

}

