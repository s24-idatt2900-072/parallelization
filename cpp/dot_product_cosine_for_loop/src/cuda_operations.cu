#include "cuda_util.h"
#include "common.h"
#include "hip/hip_fp16.h"


#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            std::cerr << "Error: " << hipGetErrorString(error) << ", file: " << __FILE__ << ", line: " << __LINE__ << std::endl; \
            exit(1); \
        } \
    } while (0)

void runCudaOperations(float* images, float* filter_real, float* filter_abs, float* output, size_t images_size, size_t images_vector_len, size_t real_vector_len, size_t filters_size, size_t output_size, unsigned int inner_len, unsigned int image_len, unsigned int filter_len) {
    float *d_images, *d_filter_real, *d_filter_abs, *d_output;
    CUDA_CHECK(hipMalloc(&d_images, images_size));
    CUDA_CHECK(hipMalloc(&d_filter_real, filters_size));
    CUDA_CHECK(hipMalloc(&d_filter_abs, filters_size));
    CUDA_CHECK(hipMalloc(&d_output, output_size));

    CUDA_CHECK(hipMemcpy(d_images, images, images_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_filter_real, filter_real, filters_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_filter_abs, filter_abs, filters_size, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(16, 16, 1);
    dim3 blocksPerGrid((image_len + 15) / 16, (filter_len + 15) / 16);

    // start measuring time of computing
    hipEvent_t start, stop;
    float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    cosineSimilarityKernel<<<blocksPerGrid, threadsPerBlock>>>(d_images, d_filter_real, d_filter_abs, d_output, inner_len, image_len, images_vector_len, real_vector_len, filter_len);
    hipDeviceSynchronize();

    // stop measuring time
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Kernel execution time: " << milliseconds << " milliseconds.\n";

    CUDA_CHECK(hipMemcpy(output, d_output, output_size, hipMemcpyDeviceToHost));

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_images);
    hipFree(d_filter_real);
    hipFree(d_filter_abs);
    hipFree(d_output);
}

void getSystemInformation() {
    int device;
    hipGetDevice(&device); // get current device ID
    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, device); // get device properties
    std::cout << "GPU Name: " << properties.name << std::endl;
    std::cout << "Compute capability: " << properties.major << "." << properties.minor << std::endl;
    std::cout << "Total global memory: " << properties.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
    std::cout << "Shared memory per block: " << properties.sharedMemPerBlock / 1024 << " KB" << std::endl;
    std::cout << "Registers per block: " << properties.regsPerBlock << std::endl;
    std::cout << "Warp size: " << properties.warpSize << std::endl;
    std::cout << "Memory Clock Rate (KHz): " << properties.memoryClockRate << std::endl;
    std::cout << "Memory Bus Width (bits): " << properties.memoryBusWidth << std::endl;
    if (properties.memoryBusWidth != 0) {
        float memoryBandwidth = 2.0 * properties.memoryClockRate * (properties.memoryBusWidth / 8) / 1.0e6;
        std::cout << "Theoretical Memory Bandwidth (GB/s): " << memoryBandwidth << std::endl;
    }
}