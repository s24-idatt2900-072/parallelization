#include <gtest/gtest.h>
#include <vector>
#include <hip/hip_runtime.h>
#include "cuda_util.h"
#include "common.h"

void fillVector(std::vector<float>& v, float value) {
    std::fill(v.begin(), v.end(), value);
}

void checkCudaError(const char* message) {
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error after %s: %s\n", message, hipGetErrorString(error));
        exit(-1);
    }
}


// Kernel wrapper to simplify testing
void runCosineSimilarityKernel(const float* images, const float* filters_real, const float* filters_abs, float* output,
                            size_t images_size, size_t image_vector_len, size_t real_vector_len, size_t filters_size, size_t output_size, unsigned int inner_len, unsigned int image_len, unsigned int filter_len) {
    // Allocate device memory
    float *d_images, *d_filters_real, *d_filters_abs, *d_output;
    hipMalloc(&d_images, images_size);
    hipMalloc(&d_filters_real, filters_size);
    hipMalloc(&d_filters_abs, filters_size);
    hipMalloc(&d_output, output_size);

    // Copy data to device
    hipMemcpy(d_images, images, images_size, hipMemcpyHostToDevice);
    hipMemcpy(d_filters_real, filters_real, filters_size, hipMemcpyHostToDevice);
    hipMemcpy(d_filters_abs, filters_abs, filters_size, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 threadsPerBlock(16, 16, 1);
    dim3 blocksPerGrid((image_len + 15) / 16, (filter_len + 15) / 16);

    // Run the kernel
    cosineSimilarityKernel<<<blocksPerGrid, threadsPerBlock>>>(d_images, d_filters_real, d_filters_abs, d_output, inner_len, image_len, image_vector_len, real_vector_len, filter_len);
    hipDeviceSynchronize();

    checkCudaError("cosineSimilarityKernel");

    // Copy the results back to host
    hipMemcpy(output, d_output, output_size, hipMemcpyDeviceToHost);
    checkCudaError("hipMemcpy");

    // Free device memory
    hipFree(d_images);
    hipFree(d_filters_real);
    hipFree(d_filters_abs);
    hipFree(d_output);
}


void runMaxPoolKernel(const float* output, float* pooled_output, size_t output_size, unsigned int pool_size, unsigned int pool_len) {
    float *d_output, *d_pooled_output;
    hipMalloc(&d_output, output_size);
    hipMalloc(&d_pooled_output, output_size);

    hipMemcpy(d_output, output, output_size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(256, 1, 1);
    dim3 blocksPeGrid((pool_len + threadsPerBlock.x - 1) / threadsPerBlock.x, 1, 1);

    maxPoolKernel<<<blocksPeGrid, threadsPerBlock>>>(d_output, d_pooled_output, pool_size, pool_len);
    hipDeviceSynchronize();

    hipMemcpy(pooled_output, d_pooled_output, pool_len*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_output);
    hipFree(d_pooled_output);
}

TEST(CosineSimilarityKernelTest, HandlesZeroInput) {
    int inner_len = 841;
    int images_len = 10;
    int filters_len = 10;
    int real_vector_len = filters_len * inner_len;

    int images_size = images_len * inner_len * sizeof(float);
    int image_vec_len = images_len * inner_len;
    int filters_size = filters_len * inner_len * sizeof(float);
    int output_size = images_len * filters_len * sizeof(float);

    // Create host vectors
    std::vector<float> h_images(images_len*inner_len, 0.0f);
    std::vector<float> h_filters_real(real_vector_len*inner_len, 0.0f);
    std::vector<float> h_filters_abs(real_vector_len*inner_len, 0.0f);
    std::vector<float> h_output(images_len*filters_len, -1.0f); // Initialize to -1 to check if it's modified

    // Run kernel
    runCosineSimilarityKernel(h_images.data(), h_filters_real.data(), h_filters_abs.data(), h_output.data(),
                              images_size, image_vec_len, real_vector_len, filters_size, output_size, inner_len, images_len, filters_len);
    

    // Check the output
    for (float val : h_output) {
        EXPECT_FLOAT_EQ(val, 0.0f);
    }
}

TEST(CosineSimilarityKernelTest, HandlesPositiveInput) {
    int inner_len = 841;
    int images_len = 10;
    int filters_len = 10;
    int real_vector_len = filters_len * inner_len;

    int images_size = images_len * inner_len * sizeof(float);
    int image_vec_len = images_len * inner_len;
    int filters_size = filters_len * inner_len * sizeof(float);
    int output_size = images_len * filters_len * sizeof(float);


    // Create host vectors
    std::vector<float> h_images(images_len*inner_len, 1.0f);
    std::vector<float> h_filters_real(real_vector_len*inner_len, 1.0f);
    std::vector<float> h_filters_abs(real_vector_len*inner_len, 1.0f);
    std::vector<float> h_output(images_len*filters_len, -1.0f); // Initialize to -1 to check if it's modified


    // Define grid and block dimensions
    dim3 blocks(16, 16, 1);
    dim3 threads((images_len + 15) / 16, (filters_len + 15) / 16);

    // Run kernel
    runCosineSimilarityKernel(h_images.data(), h_filters_real.data(), h_filters_abs.data(), h_output.data(),
                              images_size, image_vec_len, real_vector_len, filters_size, output_size, inner_len, images_len, filters_len);

    // Check the output
    for (float val : h_output) {
        EXPECT_FLOAT_EQ(val, 29.0f);
    }
}


TEST(CosineSimilarityKernelTest, HandlesRealCaseNumers) {
    int inner_len = 841;
    int image_len = 2;
    int filter_len = 2;

    size_t images_size = image_len * inner_len * sizeof(float); // alen * 841 floats
    size_t filters_size = filter_len * inner_len * sizeof(float); // blen * 841 floats
    size_t output_size = image_len * filter_len * sizeof(float); // alen * blen floats

    std::vector<float> images(image_len * inner_len, 0.0f);  // Initialize with some values
    std::vector<float> filters_real(filter_len * inner_len, 0.0f);  // Initialize with some values
    std::vector<float> filters_abs(filter_len * inner_len, 0.0f);  // Initialize with some values
    std::vector<float> output(image_len * filter_len, 0.0f);   

    int image_vec_len = image_len * inner_len;
    int real_vector_len = filter_len * inner_len;

    loadDataFromFile("mnist/mnist_padded_29x29.csv", images);
    loadDataFromFile("filters/filters_real.csv", filters_real);
    loadDataFromFile("filters/filters_abs.csv", filters_abs);


    runCosineSimilarityKernel(images.data(), filters_real.data(), filters_abs.data(), output.data(),
                              images_size, image_vec_len, real_vector_len, filters_size, output_size, inner_len, image_len, filter_len);



    const float epsilon = 3e-6f;

    EXPECT_NEAR(output[0], 0.9454422f, epsilon);
    EXPECT_NEAR(output[1], 0.94615983f, epsilon);
    EXPECT_NEAR(output[2], 0.95451612f, epsilon);
    EXPECT_NEAR(output[3], 0.95522778f, epsilon);

}


TEST(MaxPoolKernelTest, HandlesRealCaseNumers) {
    int inner_len = 841;
    int image_len = 10;
    int filter_len = 10;

    size_t images_size = image_len * inner_len * sizeof(float); // alen * 841 floats
    size_t filters_size = filter_len * inner_len * sizeof(float); // blen * 841 floats
    size_t output_size = image_len * filter_len * sizeof(float); // alen * blen floats

    std::vector<float> images(image_len * inner_len, 0.0f);  // Initialize with some values
    std::vector<float> filters_real(filter_len * inner_len, 0.0f);  // Initialize with some values
    std::vector<float> filters_abs(filter_len * inner_len, 0.0f);  // Initialize with some values
    std::vector<float> output(image_len * filter_len, 0.0f);   

    int image_vec_len = image_len * inner_len;
    int real_vector_len = filter_len * inner_len;

    loadDataFromFile("mnist/mnist_padded_29x29.csv", images);
    loadDataFromFile("filters/filters_real.csv", filters_real);
    loadDataFromFile("filters/filters_abs.csv", filters_abs);


    runCosineSimilarityKernel(images.data(), filters_real.data(), filters_abs.data(), output.data(),
                              images_size, image_vec_len, real_vector_len, filters_size, output_size, inner_len, image_len, filter_len);


    int pool_size = 5;
    int pool_len = (image_len * filter_len + pool_size - 1) / pool_size;

    std::vector<float> pooled_output(pool_len, 0.0f);

    runMaxPoolKernel(output.data(), pooled_output.data(), output_size, pool_size, pool_len);

    const float epsilon = 2e-5f;

    EXPECT_NEAR(pooled_output[0], 0.94816587f, epsilon);
    EXPECT_NEAR(pooled_output[1], 0.95112157f, epsilon);
    EXPECT_NEAR(pooled_output[2], 0.95722482f, epsilon);
    EXPECT_NEAR(pooled_output[3], 0.96015735f, epsilon);
    EXPECT_NEAR(pooled_output[4], -0.00291395f, epsilon);
    EXPECT_NEAR(pooled_output[5], -0.00245371f, epsilon);
    EXPECT_NEAR(pooled_output[6], -0.00241296f, epsilon);
    EXPECT_NEAR(pooled_output[7], -0.00190928f, epsilon);
    EXPECT_NEAR(pooled_output[8], -0.02609556f, epsilon);
    EXPECT_NEAR(pooled_output[9], -0.02077638f, epsilon);
    EXPECT_NEAR(pooled_output[10], 0.95638361f, epsilon);
    EXPECT_NEAR(pooled_output[11], 0.95942413f, epsilon);
    EXPECT_NEAR(pooled_output[12], 0.95982387f, epsilon);
    EXPECT_NEAR(pooled_output[13], 0.9626387f, epsilon);
    EXPECT_NEAR(pooled_output[14], 0.959446119f, epsilon);
    EXPECT_NEAR(pooled_output[15], 0.96225881f, epsilon);
    EXPECT_NEAR(pooled_output[16], 0.96422294f, epsilon);
    EXPECT_NEAR(pooled_output[17], 0.96686587f, epsilon);
    EXPECT_NEAR(pooled_output[18], 0.95831125f, epsilon);
    EXPECT_NEAR(pooled_output[19], 0.96106478f, epsilon);

}